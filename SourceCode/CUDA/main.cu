#include "hip/hip_runtime.h"
#include <iostream>
#include <cfloat>
#include "Sphere.hh"
#include "HitableList.hh"
#include "Camera.hh"
#include "Material.hh"

Hitable *random_scene(){
  int n = 500;
  Hitable **list = new Hitable*[n+1];
  list[0] = new Sphere(Vector3(0,-1000,0),1000, new Lambertian(Vector3(0.5,0.5,0.5)));
  int i = 1;
  for(int a = -11; a < 11; a++){
    for(int b = -11; b < 11; b++){
      float choose_mat = (rand()/(RAND_MAX + 1.0));
      Vector3 center(a+0.9*(rand()/(RAND_MAX + 1.0)), 0.2, b+0.9*(rand()/(RAND_MAX + 1.0)));
      
      if((center-Vector3(4,0.2,0)).length() > 0.9){
        if(choose_mat < 0.8){ //diffuse
          list[i++] = new Sphere(center, 0.2, new Lambertian(Vector3(
            (rand()/(RAND_MAX + 1.0))*(rand()/(RAND_MAX + 1.0)), 
            (rand()/(RAND_MAX + 1.0))*(rand()/(RAND_MAX + 1.0)), 
            (rand()/(RAND_MAX + 1.0)))));
        }
        else if(choose_mat < 0.95){ //metal
          list[i++] = new Sphere(center, 0.2, new Metal(Vector3(
            0.5*(1+(rand()/(RAND_MAX + 1.0))),
            0.5*(1+(rand()/(RAND_MAX + 1.0))),
            0.5*(1+(rand()/(RAND_MAX + 1.0)))
          ), 0.5*(rand()/(RAND_MAX + 1.0))));
        }
        else{
          list[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
        }
      }
    }
  }
  list[i++] = new Sphere(Vector3(0,1,0), 1.0, new Dielectric(1.5));
  list[i++] = new Sphere(Vector3(-4,1,0),1.0, new Lambertian(Vector3(0.4,0.2,0.1)));
  list[i++] = new Sphere(Vector3(4,1,0),1.0, new Metal(Vector3(0.7,0.6,0.5),0.0));
  
  return new HitableList(list, i);
}

Vector3 color(const Ray& ray, Hitable *world, int depth){
    hit_record rec;
    if(world->hit(ray, 0.001, MAXFLOAT, rec)){
        Ray scattered;
        Vector3 attenuation;
        if(depth < 500 && rec.mat_ptr->scatter(ray, rec, attenuation, scattered)){
            return attenuation*color(scattered, world, depth+1);
        }
        else return Vector3::Zero();
    }
    else{
        Vector3 unit_direction = unit_vector(ray.direction());
        float t = 0.5 * (unit_direction.y() + 1.0);
        return (1.0-t) * Vector3::One() + t*Vector3(0.5, 0.7, 1.0);
    }
}

int main()
{

  int nx = 2000;
  int ny = 1000;
  int ns = 1000;

  std::cout << "P3\n" << nx << " " <<  ny << "\n255" << std::endl;
  
  
  Hitable *world = random_scene();
  
  Vector3 lookfrom(13,2,3);
  Vector3 lookat(0,0,0);
  float dist_to_focus = 10.0;
  float aperture = 0.1;

  Camera cam(lookfrom, lookat, Vector3(0,1,0), 20, float(nx)/float(ny), aperture, dist_to_focus);
  
  for(int j = ny - 1; j >= 0; j--){
    for(int i = 0; i < nx; i++){
        
      Vector3 col = Vector3::Zero();
      
      for(int s = 0; s < ns; s++){
        float u = float(i + (rand()/(RAND_MAX + 1.0))) / float(nx);
        float v = float(j + (rand()/(RAND_MAX + 1.0))) / float(ny);
        
        Ray r = cam.get_ray(u, v);
        //Vector3 p = r.point_at_parameter(2.0);
        
        col += color(r, world, 0);
      }
      
      col /= float(ns);
      col = Vector3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));

      int ir = int(255.99*col[0]);
      int ig = int(255.99*col[1]);
      int ib = int(255.99*col[2]);

      std::cout << ir << " " << ig << " " << ib << std::endl;
    }
  }
}
