#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cfloat>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Sphere.cuh"
#include "MovingSphere.cuh"
#include "HitableList.cuh"
#include "Camera.cuh"
#include "Material.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
#define Random (hiprand_uniform(&local_random))

void error(const char *message) {
  
	std::cout << message << std::endl;
	exit(0);
}

void help(){

	std::cout << "\n"  << std::endl;
	std::cout << "\t[-d] [--defult] Set the parameters to default values"  << std::endl;
	std::cout << "\t                size: (2048x1080) | AAit: 10 | depth: 10 | spheres: 4 | nthreads: 32"  << std::endl;
	std::cout << "\t[-sizeX]        Size in pixels of coordinate X. Number greater than 0."  << std::endl;
	std::cout << "\t[-sizeY]        Size in pixels of coordinate Y. Number greater than 0."  << std::endl;
	std::cout << "\t[-AAit]         Number of iterations to calculate color in one pixel. Number greater than 0."  << std::endl;
	std::cout << "\t[-depth]        The attenuation of scattered ray. Number greater than 0."  << std::endl;
	std::cout << "\t[-spheres]      Factor number to calculate the number of spheres in the scene. Number greater than 0." << std::endl;
	std::cout << "\t[-nthreads]     Number of threads to use" << std::endl;
	std::cout << "\t[-nGPUs]        Number of GPUs to distribute the work" << std::endl;
	std::cout << "\t[-f][--file]    File name of pic generated." << std::endl;
	std::cout << "\t[-h][--help]    Show help." << std::endl;
	std::cout << "\t                #spheres = (2*spheres)*(2*spheres) + 4" << std::endl;
	std::cout << "\n" << std::endl;
	std::cout << "Examples of usage:" << std::endl;
	std::cout << "./path_tracing_NGPUs -d"  << std::endl;
	std::cout << "./path_tracing_NGPUs -nthreads 16 -sizeX 2000"<< std::endl;
	exit(0);
  
}

void parse_argv(int argc, char **argv, int &nx, int &ny, int &ns, int &depth, int &dist, int &nthreads, std::string &filename, int &numGPUs,const int count){
  
	if(argc <= 1) error("Error usage. Use [-h] [--help] to see the usage.");
  
	nx = 2048; ny = 1080; ns = 10; depth = 10; dist = 4; nthreads = 32; filename = "pic.ppm"; numGPUs = 1;
  
	bool v_default = false;
  
	for(int i = 1; i < argc; i += 2){
    
		if(v_default) error("Error usage. Use [-h] [--help] to see the usage.");
		
		if (std::string(argv[i]) == "-d" || std::string(argv[i]) == "--default"){
			if((i+1) < argc) error("The default parameter cannot have more arguments.");
			std::cerr << "Default\n";
			v_default = true;
		}
		else if (std::string(argv[i]) == "-sizeX"){
			if((i+1) >= argc) error("-sizeX value expected");
			nx = atoi(argv[i+1]);
			if(nx == 0) error("-sizeX value expected or cannot be 0");
		}
		else if(std::string(argv[i]) == "-sizeY"){
			if((i+1) >= argc) error("-sizeY value expected");
			ny = atoi(argv[i+1]);
			if(ny == 0) error("-sizeY value expected or cannot be 0");
		}
		else if(std::string(argv[i]) == "-AAit"){
			if((i+1) >= argc) error("-AAit value expected");
			ns = atoi(argv[i+1]);
			if(ns == 0) error("-AAit value expected or cannot be 0");
		}
		else if(std::string(argv[i]) == "-depth"){
			if((i+1) >= argc) error("-depth value expected");
			depth = atoi(argv[i+1]);
			if(depth == 0) error("-depth value expected or cannot be 0");
		}
		else if(std::string(argv[i]) == "-spheres"){
			if((i+1) >= argc) error("-spheres value expected");
			dist = atoi(argv[i+1]);
			if(dist == 0) error("-spheres value expected or cannot be 0");
		}
		else if(std::string(argv[i]) == "-nthreads"){
			if((i+1) >= argc) error("-nthreads value expected");
			nthreads = atoi(argv[i+1]);
			if(nthreads == 0) error("-nthreads value expected or cannot be 0");
		}
		else if(std::string(argv[i]) == "-f" || std::string(argv[i]) == "--file"){
			if((i+1) >= argc) error("-name file expected");
			filename = std::string(argv[i+1]);
			filename = filename+".ppm";
		}
		else if(std::string(argv[i]) == "-nGPUs"){
			if((i+1) >= argc) error("-nGPUs value expected");
			numGPUs = atoi(argv[i+1]);
			if(numGPUs == 0) error("-nGPUs value expected or cannot be 0");
			numGPUs = std::min(numGPUs, count);
		}
		else if(std::string(argv[i]) == "-h" || std::string(argv[i]) == "--help" ){
			help();
		}
		else{
			error("Error usage. Use [-h] [--help] to see the usage.");
		}
	}
}

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line){
	if(result){
		std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << std::endl;
		std::cout << hipGetErrorString(result) << std::endl;
		hipDeviceReset();
		exit(99);
	}
}

__global__ void free_world(Hitable **d_list, Hitable **d_world, Camera **d_cam) {
	
	int n = (*d_world)->length();
	for(int i = 0; i < n; i++){
		delete ((Sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_cam;
	
}

__global__ void create_world(Hitable **d_list, Hitable **d_world, Camera **d_cam, int nx, int ny, int dist, hiprandState *random){
  
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		hiprandState local_random = *random;

		d_list[0] = new Sphere(Vector3(0,-1000,-1), 1000, new Lambertian(Vector3(0.5, 0.5, 0.5)));
    
		int i = 1;
		for (int a = -dist; a < dist; a++) {
			for (int b = -dist; b < dist; b++) {
				float material = Random;
				Vector3 center(a+0.9*Random, 0.2, b+0.9*Random);
	
				if ((center-Vector3(0,0,0)).length() > 0.995) {
					if (material < 0.8) d_list[i++] = new MovingSphere(center, center+Vector3(0,0.5*Random,0),0.0,1.0,.2,new Lambertian(Vector3(Random*Random, Random*Random, Random*Random)));
					else if (material < 0.95) d_list[i++] = new Sphere(center, 0.2, new Metal(Vector3(0.5*(1.0+Random), 0.5*(1.0+Random), 0.5*(1.0+Random)),0.5*Random));
					else d_list[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
				}
			}
		}
	
		d_list[i++] = new Sphere(Vector3( 0, 1, 0), 1.0, new Dielectric(1.5));
		d_list[i++] = new Sphere(Vector3(-4, 1, 0), 1.0, new Lambertian(Vector3(0.4, 0.2, 0.1)));
		d_list[i++] = new Sphere(Vector3( 4, 1, 0), 1.0, new Metal(Vector3(0.7, 0.6, 0.5),0.0));
		
		d_list[i++] = new Sphere(Vector3( 4, 1, 5), 1.0, new Metal(Vector3(0.9, 0.2, 0.2),0.0));
	
		*random = local_random;
		
		*d_world = new HitableList(d_list,i);
		
		Vector3 lookfrom(13,2,3);
		Vector3 lookat(0,0,0);
		Vector3 up(0,1,0);
		float dist_to_focus = 10; (lookfrom-lookat).length();
		float aperture = 0.1;
		*d_cam = new Camera(lookfrom, lookat, up, 20, float(nx)/float(ny), aperture, dist_to_focus,0.0,0.1);
	}
}

__device__ Vector3 color(const Ray& ray, Hitable **world, int depth, hiprandState *random){
  
	Ray cur_ray = ray;
	Vector3 cur_attenuation = Vector3(1.0,1.0,1.0);
	for(int i = 0; i < depth; i++){ 
		hit_record rec;
		if( (*world)->hit(cur_ray, 0.001, FLT_MAX, rec)) {
      
			Ray scattered;
			Vector3 attenuation;
			if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, random)){
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else return Vector3(0.0,0.0,0.0);
		}
		else {
			Vector3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5*(unit_direction.y() + 1.0);
			Vector3 c = (1.0 - t)*Vector3(1.0,1.0,1.0) + t*Vector3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	
	return Vector3(0.0,0.0,0.0);
	
}

__global__ void rand_init(hiprandState *random, int seed) {
  
	if(threadIdx.x == 0 && blockIdx.x == 0){
		hiprand_init(seed, 0, 0, random);
	}
	
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state,unsigned long long seed) {
  
  int num = blockIdx.x*blockDim.x + threadIdx.x;
  
  int i = num%max_x;
  int j = num/max_x;
  
  if( (i >= max_x) || (j >= max_y) ) return;
    
  int pixel_index = num;
    
  hiprand_init((seed << 20) + pixel_index, 0, 0, &rand_state[pixel_index]);
  
}

__global__ void render(Vector3 *fb, int max_x, int max_y, int ns, Camera **cam, Hitable **world, hiprandState *d_rand_state, int depth) {
	
	int num = blockIdx.x*blockDim.x + threadIdx.x;
 
	int i = num%max_x;
	int j = num/max_x;
  
	hiprandState local_random;
  
	int pixel_index = num;
    
	local_random = d_rand_state[pixel_index];
    
	Vector3 col(0,0,0);
    
	for(int s = 0; s < ns; s++){
    
		float u = float(i + Random) / float(max_x);
		float v = float(j + Random) / float(max_y);
      
		Ray r = (*cam)->get_ray(u, v, &local_random);
		col += color(r, world, depth, &local_random);
      
	}
    
	d_rand_state[pixel_index] = local_random;
    
	col /= float(ns);
    
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    
    fb[pixel_index] = col;
    
}

int main(int argc, char **argv) {
  
	std::cout << "GPU Info " << std::endl;
  
    hipSetDevice(0);
    int device;
    hipGetDevice(&device);
    
    hipDeviceProp_t properties;
    checkCudaErrors( hipGetDeviceProperties( &properties, device ) );
    
    if( properties.major > 3 || ( properties.major == 3 && properties.minor >= 5 ) )
    {
      
		std::cout << "Running on GPU " << device << " (" << properties.name << ")" << std::endl;
		std::cout << "Compute mode: " << properties.computeMode << std::endl;
		std::cout << "Concurrent Kernels: " << properties.concurrentKernels << std::endl;
		std::cout << "Warp size: " << properties.warpSize << std::endl;
		std::cout << "Major: " << properties.major << " Minor: " << properties.minor << "\n\n" << std::endl;
      
    }
    else std::cout << "GPU " << device << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;

  
	hipEvent_t E0, E1;
	hipEventCreate(&E0); hipEventCreate(&E1);
  
	float totalTime;
  
	int nx, ny, ns, depth, dist, nthreads, numGPUs;
	std::string filename;
  
	parse_argv(argc, argv, nx, ny, ns, depth, dist, nthreads, filename, numGPUs, 1);

	int n = (2*dist)*(2*dist)+5;
  
	std::cout << "Creating " << filename << " with (" << nx << "," << ny << ") pixels with " << nthreads << " threads, using " << numGPUs << " GPUs." << std::endl;
	std::cout << "With " << ns << " iterations for AntiAliasing and depth of " << depth << "." << std::endl;
	std::cout << "The world have " << n << " spheres." << std::endl;

	/* Seed for CUDA Random */
	unsigned long long int seed = 1000;
  
	/* #pixels of the image */
	int num_pixels = nx*ny;
	
	/* Host variables */
	float fb_size = num_pixels*sizeof(Vector3);
	float drand_size = num_pixels*sizeof(hiprandState);
	Vector3 *h_frameBuffer;
  
	int blocks = (nx * ny)/(numGPUs * nthreads);
	
	/* Allocate Memory Host */
	hipHostMalloc((Vector3**)&h_frameBuffer, fb_size);
	
	/* Device variables */
	Vector3 *d_frameBuffer;
	Hitable **d_list;
	Hitable **d_world;
	Camera **d_cam;
	hiprandState *d_rand_state;
	hiprandState *d_rand_state2;
  
	/* Allocate memory on device */
	hipMallocManaged((void **)&d_frameBuffer, fb_size);
	hipMalloc((void **)&d_list, n*sizeof(Hitable *));
	hipMalloc((void **)&d_world, sizeof(Hitable *));
	hipMalloc((void **)&d_cam, sizeof(Camera *));
	hipMalloc((void **)&d_rand_state, drand_size);
	hipMalloc((void **)&d_rand_state2, sizeof(hiprandState));
  
	hipEventRecord(E0,0);
	hipEventSynchronize(E0);

	rand_init<<<1,1>>>(d_rand_state2, seed);
	checkCudaErrors(hipGetLastError());
	create_world<<<1,1>>>(d_list, d_world, d_cam, nx, ny, dist, d_rand_state2);
	checkCudaErrors(hipGetLastError());
	render_init<<<blocks, nthreads>>>(nx, ny, d_rand_state, seed);
	checkCudaErrors(hipGetLastError());
	render<<<blocks, nthreads>>>(d_frameBuffer, nx, ny, ns, d_cam, d_world, d_rand_state, depth);
	checkCudaErrors(hipGetLastError());
	
	hipMemcpy(h_frameBuffer, d_frameBuffer, fb_size, hipMemcpyDeviceToHost);
	checkCudaErrors(hipGetLastError());
  
	hipEventRecord(E1,0);
	hipEventSynchronize(E1);
  
	hipEventElapsedTime(&totalTime,E0,E1);
  
	checkCudaErrors(hipGetLastError());
  
	std::cout << "Total time: " << totalTime << " milisegs. " << std::endl;
  
	std::cout << "Generating file image..." << std::endl;
	std::ofstream pic;
	pic.open(filename.c_str());
  
	pic << "P3\n" << nx << " " << ny << "\n255\n";
  
	for(int j = ny-1; j >= 0; j--){
		for(int i = 0; i < nx; i++){

			size_t pixel_index = j*nx + i;
      
			Vector3 col = h_frameBuffer[pixel_index];
      
			int ir = int(255.99*col.r());
			int ig = int(255.99*col.g());
			int ib = int(255.99*col.b());
			
			pic << ir << " " << ig << " " << ib << "\n";
		}
	}
  
	pic.close();
  
	free_world<<<1,1>>>(d_list,d_world,d_cam);
	hipFree(d_cam);
	hipFree(d_world);
	hipFree(d_list);
	hipFree(d_rand_state);
	hipFree(d_frameBuffer);
	
	hipEventDestroy(E0); hipEventDestroy(E1);
	
	hipDeviceReset();  
}
