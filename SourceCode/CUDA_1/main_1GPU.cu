#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <cfloat>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Camera.cuh"
#include "Scene.cuh"
#include "HitableList.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
#define Random (hiprand_uniform(&local_random))

void error(const char *message) {
  
  std::cout << message << std::endl;
  exit(0);
}

void format() {
  std::cout << "File format for scene." << std::endl;
  std::cout << "\t #          Comment, skip line." << std::endl;
  std::cout << "Spheres -> type center material" << std::endl;
  std::cout << "\t 1          Indicates that the 3D model is a Sphere object." << std::endl;
  std::cout << "\t Center     The center of the Sphere." << std::endl;
  std::cout << "\t Radius     The radius of the Sphere." << std::endl;
  std::cout << "\t Material -> type albedo [fuzz] [ref_idx]" << std::endl;
  std::cout << "\t\t 0        LAMBERTIAN" << std::endl;
  std::cout << "\t\t 1        METAL" << std::endl;
  std::cout << "\t\t 2        DIELECTRIC" << std::endl;
  std::cout << "\t\t 3        DIFFUSE LIGHT" << std::endl;
  std::cout << "\t\t albedo   Defines the color." << std::endl;
  std::cout << "\t\t fuzz     Only for METAL." << std::endl;
  std::cout << "\t\t ref_idx  Only for DIELECTRIC." << std::endl;
  std::cout << "Examples of declaration:\n" << std::endl;
  std::cout << "# my scene" << std::endl;
  std::cout << "Object   Center Rad Material  Albedo        Fuzz/ref_idx" << std::endl;
  std::cout << "1       0 1 0   2   1         0.5 0.78 0.9        " << std::endl;
  std::cout << "1       0 4 0   2   2         1   0    0.9    2   " << std::endl;
  std::cout << "1       1 4 1   2   3         0.9 0.9  0.9    1.5 " << std::endl;
}

void help(){

  std::cout << "\n"  << std::endl;
  std::cout << "\t[-d] [--defult] Set the parameters to default values"  << std::endl;
  std::cout << "\t                size: (2048x1080) | AAit: 10 | depth: 10 | spheres: 4 | nthreads: 32"  << std::endl;
  std::cout << "\t[-sizeX]        Size in pixels of coordinate X. Number greater than 0."  << std::endl;
  std::cout << "\t[-sizeY]        Size in pixels of coordinate Y. Number greater than 0."  << std::endl;
  std::cout << "\t[-AAit]         Number of iterations to calculate color in one pixel. Number greater than 0."  << std::endl;
  std::cout << "\t[-depth]        The attenuation of scattered ray. Number greater than 0."  << std::endl;
  std::cout << "\t[-spheres]      Factor number to calculate the number of spheres in the scene. Number greater than 0." << std::endl;
  std::cout << "\t[-light]        Turn on/off the ambient light. Values can be ON/OFF" << std::endl;
  std::cout << "\t[-nthreads]     Number of threads to use" << std::endl;
  std::cout << "\t[-nGPUs]        Number of GPUs to distribute the work" << std::endl;
  std::cout << "\t[-i][--image]   File name of pic generated." << std::endl;
  std::cout << "\t[-f][--file]    File name of the scene." << std::endl;
  std::cout << "\t[-h][--help]    Show help." << std::endl;
  std::cout << "\t                #spheres = (2*spheres)*(2*spheres) + 4" << std::endl;
  std::cout << "\n" << std::endl;
  std::cout << "Examples of usage:" << std::endl;
  std::cout << "./path_tracing_NGPUs -d"  << std::endl;
  std::cout << "./path_tracing_NGPUs -nthreads 16 -sizeX 2000"<< std::endl;
  format();
  exit(1);
  
}

void parse_argv(int argc, char **argv, int &nx, int &ny, int &ns, int &depth, int &dist, int &nthreads, std::string &image, std::string &filename, int &numGPUs, bool &light, bool &random,const int count){
  
  if(argc <= 1) error("Error usage. Use [-h] [--help] to see the usage.");
  
  nx = 1280; ny = 720; ns = 50; depth = 50; dist = 11; image = "random"; light = true; random = true;
  
  nthreads = 32; numGPUs = 1;
  
  bool v_default = false;
  
  for(int i = 1; i < argc; i += 2){
    
    if(v_default) error("Error usage. Use [-h] [--help] to see the usage.");
    
    if (std::string(argv[i]) == "-d" || std::string(argv[i]) == "--default"){
      if((i+1) < argc) error("The default parameter cannot have more arguments.");
      std::cerr << "Default\n";
      v_default = true;
    }
    else if (std::string(argv[i]) == "-sizeX"){
      if((i+1) >= argc) error("-sizeX value expected");
      nx = atoi(argv[i+1]);
      if(nx == 0) error("-sizeX value expected or cannot be 0");
    }
    else if(std::string(argv[i]) == "-sizeY"){
      if((i+1) >= argc) error("-sizeY value expected");
      ny = atoi(argv[i+1]);
      if(ny == 0) error("-sizeY value expected or cannot be 0");
    }
    else if(std::string(argv[i]) == "-AAit"){
      if((i+1) >= argc) error("-AAit value expected");
      ns = atoi(argv[i+1]);
      if(ns == 0) error("-AAit value expected or cannot be 0");
    }
    else if(std::string(argv[i]) == "-depth"){
      if((i+1) >= argc) error("-depth value expected");
      depth = atoi(argv[i+1]);
      if(depth == 0) error("-depth value expected or cannot be 0");
    }
    else if(std::string(argv[i]) == "-spheres"){
      if((i+1) >= argc) error("-spheres value expected");
      dist = atoi(argv[i+1]);
      if(dist == 0) error("-spheres value expected or cannot be 0");
    }
    else if(std::string(argv[i]) == "-nthreads"){
      if((i+1) >= argc) error("-nthreads value expected");
      nthreads = atoi(argv[i+1]);
      if(nthreads == 0) error("-nthreads value expected or cannot be 0");
    }
    else if(std::string(argv[i]) == "-i" || std::string(argv[i]) == "--image"){
      if((i+1) >= argc) error("--image / -i file expected");
      filename = std::string(argv[i+1]);
    }
    else if(std::string(argv[i]) == "-f" || std::string(argv[i]) == "--file"){
      if((i+1) >= argc) error("-name file expected");
      filename = std::string(argv[i+1]);
      image = filename;
      filename = filename+".txt";
      random = false;
    }
    else if(std::string(argv[i]) == "-nGPUs"){
      if((i+1) >= argc) error("-nGPUs value expected");
      numGPUs = atoi(argv[i+1]);
      if(numGPUs == 0) error("-nGPUs value expected or cannot be 0");
      numGPUs = std::min(numGPUs, count);
    }
    else if(std::string(argv[i]) == "-light") {
      if((i+1) >= argc) error("-light value expected");
      if(std::string(argv[i+1]) == "ON") light = true;
      else if(std::string(argv[i+1]) == "OFF") light = false;
    }
    else if(std::string(argv[i]) == "-h" || std::string(argv[i]) == "--help" ){
      help();
    }
    else{
      error("Error usage. Use [-h] [--help] to see the usage.");
    }
  }
  if(!light) image = image+"_noktem";
  image = image+".ppm";
}

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line){
  if(result){
    std::cout << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << std::endl;
    std::cout << hipGetErrorString(result) << std::endl;
    hipDeviceReset();
    exit(99);
  }
}

void properties(){
    
  std::cout << "GPU Info " << std::endl;

  hipSetDevice(0);
  int device;
  hipGetDevice(&device);

  hipDeviceProp_t properties;
  checkCudaErrors( hipDeviceSetLimit( hipLimitMallocHeapSize, 67108864 ) );
  checkCudaErrors( hipDeviceSetLimit( hipLimitStackSize, 131072 ) );
  checkCudaErrors( hipGetDeviceProperties( &properties, device ) );

  size_t limit1;
  checkCudaErrors( hipDeviceGetLimit( &limit1, hipLimitMallocHeapSize ) );
  size_t limit2;
  checkCudaErrors( hipDeviceGetLimit( &limit2, hipLimitStackSize ) );

  if( properties.major > 3 || ( properties.major == 3 && properties.minor >= 5 ) ) {
    std::cout << "Running on GPU " << device << " (" << properties.name << ")" << std::endl;
    std::cout << "Compute mode: " << properties.computeMode << std::endl;
    std::cout << "Concurrent Kernels: " << properties.concurrentKernels << std::endl;
    std::cout << "Warp size: " << properties.warpSize << std::endl;
    std::cout << "Major: " << properties.major << " Minor: " << properties.minor << std::endl;
    std::cout << "Cuda limit heap size: " << limit1 << std::endl;
    std::cout << "Cuda limit stack size: " << limit2 << "\n\n" << std::endl;
  }
  else std::cout << "GPU " << device << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
}

__device__ Vector3 color(const Ray& ray, HitableList **d_world, int depth, bool light, hiprandState *random){
  
  Ray cur_ray = ray;
  Vector3 cur_attenuation = Vector3(1.0,1.0,1.0);
  for(int i = 0; i < depth; i++){ 
    hit_record rec;
    if( (*d_world)->checkCollision(cur_ray, 0.001, FLT_MAX, rec)) {
      Ray scattered;
      Vector3 attenuation;
      Vector3 emitted = rec.mat_ptr.emitted();
      if(rec.mat_ptr.scatter(cur_ray, rec, attenuation, scattered, random)){
        cur_attenuation *= attenuation;
        cur_attenuation += emitted;
        cur_ray = scattered;
      }
      else return cur_attenuation * emitted;
    }
    else {
      if(light) {
        Vector3 unit_direction = unit_vector(cur_ray.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
        Vector3 c = (1.0 - t)*Vector3::One() + t*Vector3(0.5, 0.7, 1.0);
        return cur_attenuation * c;
      }
      else return Vector3::Zero();
    }
  }
  return Vector3::Zero();
}

__global__ void setUpCameraWorld(Camera **d_cam, int nx, int ny, HitableList **d_world, Sphere *d_objects, int size) {
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    Vector3 lookfrom(13,2,3);
    Vector3 lookat(0,0,0);
    Vector3 up(0,1,0);
    float dist_to_focus = 10.0;
    float aperture = 0.1;
    *d_cam = new Camera(lookfrom, lookat, up, 20, float(nx)/float(ny), aperture, dist_to_focus,0.0,0.1);
    
    *d_world = new HitableList(d_objects,size);
  }
}

__global__ void rand_init(hiprandState *random, int seed) {
  
  if(threadIdx.x == 0 && blockIdx.x == 0) {
    hiprand_init(seed, 0, 0, random);
  }

}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state,unsigned long long seed) {
  
  int num = blockIdx.x*blockDim.x + threadIdx.x;
  
  int i = num%max_x;
  int j = num/max_x;
  
  if( (i >= max_x) || (j >= max_y) ) return;
    
  int pixel_index = num;
    
  hiprand_init((seed << 20) + pixel_index, 0, 0, &rand_state[pixel_index]);
  
}

__global__ void render(Vector3 *fb, int max_x, int max_y, int ns, Camera **cam, HitableList **d_world, hiprandState *d_rand_state, int depth, bool light) {

  int num = blockIdx.x*blockDim.x + threadIdx.x;
 
  int i = num%max_x;
  int j = num/max_x;

  hiprandState local_random;

  int pixel_index = num;
    
  local_random = d_rand_state[pixel_index];
    
  Vector3 col(0,0,0);
    
  for(int s = 0; s < ns; s++){

    float u = float(i + Random) / float(max_x);
    float v = float(j + Random) / float(max_y);
      
    Ray r = (*cam)->get_ray(u, v, &local_random);
    col += color(r, d_world, depth, light, &local_random);
    
  }

  d_rand_state[pixel_index] = local_random;
    
  col /= float(ns);

  col[0] = sqrt(col[0]);
  col[1] = sqrt(col[1]);
  col[2] = sqrt(col[2]);

  fb[pixel_index] = col;
    
}

int main(int argc, char **argv) {
  
  hipDeviceReset();
  
  properties();

  hipEvent_t E0, E1;
  hipEventCreate(&E0); 
  hipEventCreate(&E1);
  
  float totalTime;

  int nx, ny, ns, depth, dist, nthreads, numGPUs;
  bool light, random;
  std::string filename, image;

  parse_argv(argc, argv, nx, ny, ns, depth, dist, nthreads, image, filename, numGPUs, light, random, 1);

  /* Seed for CUDA Random */
  unsigned long long int seed = 1000;

  /* #pixels of the image */
  int num_pixels = nx*ny;
  int size = 0;

  /* Host variables */
  float fb_size = num_pixels*sizeof(Vector3);
  float drand_size = num_pixels*sizeof(hiprandState);
  float cam_size = sizeof(Camera*);
  float world_size = sizeof(HitableList*);
  Vector3 *h_frameBuffer;

  int blocks = (nx * ny)/(numGPUs * nthreads);

  /* Create world */
  Scene scene(dist);
  if(random) scene.loadScene(RANDOM);
  else scene.loadScene(FFILE,filename);
  
  size = scene.getSize();
  float ob_size = size*sizeof(Sphere);
  
  std::cout << "\nCreating " << image << " with (" << nx << "," << ny << ") pixels with " << nthreads << " threads, using " << numGPUs << " GPUs." << std::endl;
  std::cout << "With " << ns << " iterations for AntiAliasing and depth of " << depth << "." << std::endl;
  std::cout << "The world have " << size << " spheres." << std::endl;
  if(light) std::cout << "Ambient light ON" << std::endl;
  else std::cout << "Ambient light OFF" << std::endl;

  /* Device variables */
  Vector3 *d_frameBuffer;
  Sphere *d_objects;
  Camera **d_cam;
  HitableList **d_world;
  hiprandState *d_rand_state;
  
  /* Allocate Memory Host */
  hipHostMalloc((Vector3**)&h_frameBuffer, fb_size);

  /* Allocate memory on device */
  hipMallocManaged((void **)&d_frameBuffer, fb_size);
  hipMalloc((void **)&d_objects, ob_size);
  hipMalloc((void **)&d_world, world_size);
  hipMalloc((void **)&d_cam, cam_size);
  hipMalloc((void **)&d_rand_state, drand_size);
  
  hipEventRecord(E0,0);
  hipEventSynchronize(E0);
  
  hipMemcpy(d_objects, scene.getObjects(), ob_size, hipMemcpyHostToDevice);
  checkCudaErrors(hipGetLastError());
  
  setUpCameraWorld<<<1,1>>>(d_cam, nx, ny, d_world, d_objects, size);
  checkCudaErrors( hipGetLastError() );
  
  render_init<<<blocks, nthreads>>>(nx, ny, d_rand_state, seed);
  checkCudaErrors(hipGetLastError());
  
  render<<<blocks, nthreads>>>(d_frameBuffer, nx, ny, ns, d_cam, d_world, d_rand_state, depth, light);
  checkCudaErrors(hipGetLastError());

  hipMemcpy(h_frameBuffer, d_frameBuffer, fb_size, hipMemcpyDeviceToHost);
  checkCudaErrors(hipGetLastError());

  hipEventRecord(E1,0);
  hipEventSynchronize(E1);

  hipEventElapsedTime(&totalTime,E0,E1);

  checkCudaErrors(hipGetLastError());
  
  std::cout << "Total time: " << totalTime << " milisegs. " << std::endl;

  std::cout << "Generating file image..." << std::endl;
  std::ofstream pic;
  pic.open(image.c_str());

  pic << "P3\n" << nx << " " << ny << "\n255\n";
  
  for(int j = ny-1; j >= 0; j--){
    for(int i = 0; i < nx; i++){

      size_t pixel_index = j*nx + i;

      Vector3 col = h_frameBuffer[pixel_index];

      int ir = int(255.99*col.r());
      int ig = int(255.99*col.g());
      int ib = int(255.99*col.b());

      pic << ir << " " << ig << " " << ib << "\n";
    }
  }
  
  pic.close();

  hipFree(d_cam);
  hipFree(d_world);
  hipFree(d_objects);
  hipFree(d_rand_state);
  hipFree(d_frameBuffer);

  hipEventDestroy(E0); 
  hipEventDestroy(E1);
}
