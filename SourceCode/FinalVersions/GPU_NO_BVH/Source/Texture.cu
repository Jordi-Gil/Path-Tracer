#include "Texture.cuh"

__host__ __device__ Texture::Texture(int t, const Vector3 &a, unsigned char *data, int sx, int sy, int _textureIndex, bool _fH, bool _fV, bool _flipUV) {
  
  type = t;
  albedo = a;
  h_image = data;
  nx = sx;
  ny = sy;
  flipHorizontal = _fH;
  flipVertical = _fV;
  flipUV = _flipUV;
  textureIndex = _textureIndex;
}

__host__ __device__ Vector3 Texture::imValue(float u, float v, bool oneTex, unsigned char **d_textures){
  
  u = flipHorizontal ? 1-u : u; 
  v = flipVertical ? 1-v : v;
  
  if(flipUV) {
    float aux = u;
    u = v;
    v = aux;
  }
  
  int i = u * (nx-1);
  int j = v * (ny-1);
  
  if(i < 0) i = 0;
  if(j < 0) j = 0;
  
  if(i > nx-1) i = nx-1;
  if(j > ny-1) j = ny-1;
  
  float r;
  float g;
  float b;
  
  if(!oneTex or textureIndex == 999){
    r = int(d_image[3*i*nx + 3*j + 0]) / 255.0f;
    g = int(d_image[3*i*nx + 3*j + 1]) / 255.0f;
    b = int(d_image[3*i*nx + 3*j + 2]) / 255.0f;
  }
  else{
    unsigned char *image = d_textures[textureIndex];
    r = int(image[3*i*nx + 3*j + 0]) / 255.0f;
    g = int(image[3*i*nx + 3*j + 1]) / 255.0f;
    b = int(image[3*i*nx + 3*j + 2]) / 255.0f;
  }
  
  return Vector3(r,g,b);

}

__host__ __device__ Vector3 Texture::value(float u, float v, bool oneTex, unsigned char **d_textures){
  
  if(type == CONSTANT) return albedo;
  else return imValue(u, v, oneTex, d_textures);
  
}

__host__ void Texture::hostToDevice(int numGPUs){
  
  if(type == IMAGE){
  
		hipSetDevice(numGPUs);
		
    float size = sizeof(unsigned char) * nx * ny * 3;
    hipMalloc((void **)&d_image, size);
    assert(hipMemcpy(d_image, h_image, size, hipMemcpyHostToDevice) == hipSuccess);
  }
}
