#include "hip/hip_runtime.h"
#include "Sphere.cuh"

__device__ void get_sphere_uv(const Vector3& p, float& u, float& v) {
    float phi = atan2(p.z(), p.x());
    float theta = asin(p.y());
    u = 1-(phi + M_PI) / (2*M_PI);
    v = (theta + M_PI/2) / M_PI;
}

__host__ __device__ Sphere::Sphere(Vector3 cen, float r, Material mat) { 
  center = cen;
  radius = r;
  mat_ptr = mat;
  morton_code = 0;
  bounding_box(box);
}

__device__ bool Sphere::hit(const Ray& r, float t_min, float t_max, hit_record &rec) {
  
  Vector3 oc = r.origin() - center;
  float a = dot(r.direction(), r.direction());
  float b = dot(oc, r.direction());
  float c = dot(oc, oc) - radius*radius;
  float discriminant = b*b - a*c;
  
  if(discriminant > 0){
    float temp = (-b - sqrt(b*b-a*c))/a;
    if(temp < t_max && temp > t_min){
      rec.t = temp;
      get_sphere_uv((rec.t-center)/radius, rec.u, rec.v);
      rec.point = r.point_at_parameter(rec.t);
      rec.normal = (rec.point - center) / radius;
      rec.mat_ptr = this->mat_ptr;
      return true;
    }
    
    temp = (-b + sqrt(b*b-a*c))/a;
    if(temp < t_max && temp > t_min){
      rec.t = temp;
      get_sphere_uv((rec.t-center)/radius, rec.u, rec.v);
      rec.point = r.point_at_parameter(rec.t);
      rec.normal = (rec.point - center) / radius;
      rec.mat_ptr = this->mat_ptr;
      return true;
    }
  }
  return false;
}

__host__ __device__ void Sphere::bounding_box(aabb& box) {
  
  box = aabb(center - Vector3(radius), center + Vector3(radius));
  
}

__host__ __device__ aabb Sphere::getBox() {
    return box;
}

__host__ __device__ long long Sphere::getMorton() {
    return morton_code;
}

__host__ __device__ void Sphere::setMorton(long long code) {
    morton_code = code;
}

__host__ __device__ Vector3 Sphere::getCenter() {
    return center;
}

__host__ __device__ float Sphere::getRadius() {
    return radius;
}

__host__ __device__ Material Sphere::getMaterial() {
    return mat_ptr;
}

