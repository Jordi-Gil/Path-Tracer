#include "hip/hip_runtime.h"
#include "Triangle.cuh"

__host__ __device__ Triangle::Triangle(Vector3 v1, Vector3 v2, Vector3 v3, Material mat, Vector3 uv1, Vector3 uv2, Vector3 uv3) {
    vertex[0] = v1;
    vertex[1] = v2;
    vertex[2] = v3;
    centroid = (v1+v2+v3)/3;
    mat_ptr = mat;
    morton_code = 0;
    bounding_box(box);
    uv[0] = uv1;
    uv[1] = uv2;
    uv[2] = uv3;
    area = (cross(v2-v1, v3-v1)).length()/2;
}

__host__ __device__ bool Triangle::hit(const Ray& r, float t_min, float t_max, hit_record& rec) {
  
  Vector3 e1 = vertex[1] - vertex[0];
  Vector3 e2 = vertex[2] - vertex[0];
  
  Vector3 P = cross(r.direction(), e2);
  float determinant = dot(e1, P);
  
  if(determinant > -t_min and determinant < t_min) 
    return false;
  float invDet = 1.0f / determinant;
  
  Vector3 T = r.origin() - vertex[0];
  float u = dot(T, P) * invDet;
  
  if(u < 0.0f || u > 1.0f) return false;
  
  Vector3 Q = cross(T, e1);
  float v = dot(r.direction(), Q) * invDet;
  
  if(v < 0.0f || u + v > 1.0f) return false;
  
  float temp = dot(e2, Q) * invDet;
  if(temp > t_min && temp < t_max) {
    rec.t = temp;
    Vector3 aux = (1-u-v)*uv[0] + u*uv[1] + v*uv[2];
    rec.u = aux[0];
    rec.v = aux[1];
    rec.point = r.point_at_parameter(rec.t);
    rec.normal = normalize(cross(e1, e2));
    rec.mat_ptr = this->mat_ptr;
    
    return true;
  }
  
  return false;
}

__device__ float Triangle::pdf_value(const Vector3 &origin, const Vector3 &direction) {
  
  hit_record rec;
  if(this->hit(Ray(origin, direction), 0.001, FLT_MAX, rec)){
    float distance = rec.t * rec.t * direction.squared_length();
    float cosine = dot(direction, rec.normal);
    return (distance/(cosine*area));
  }
  return 0;
}

__device__ Vector3 Triangle::random(const Vector3 &origin, hiprandState *random) {
  
  float r1 = hiprand_uniform(random);
  float r2 = hiprand_uniform(random);
  float sr1 = sqrt(r1);
  
  Vector3 random_point((1.0 - sr1) * vertex[0] + sr1 * (1.0 - r2) * vertex[1] + sr1 * r2 * vertex[2]);
  
  return(random_point - origin);
}

__host__ __device__ void Triangle::bounding_box(aabb& box) {
  
	float x_max = math::max(math::max(vertex[0].x(),vertex[1].x()),vertex[2].x());
	float y_max = math::max(math::max(vertex[0].y(),vertex[1].y()),vertex[2].y());
	float z_max = math::max(math::max(vertex[0].z(),vertex[1].z()),vertex[2].z());
	
	float x_min = math::min(math::min(vertex[0].x(),vertex[1].x()),vertex[2].x());
	float y_min = math::min(math::min(vertex[0].y(),vertex[1].y()),vertex[2].y());
	float z_min = math::min(math::min(vertex[0].z(),vertex[1].z()),vertex[2].z());
  
  if(x_max == x_min) { x_max += 0.0005; x_min -= 0.0005; }
  if(y_max == y_min) { y_max += 0.0005; y_min -= 0.0005; }
  if(z_max == z_min) { z_max += 0.0005; z_min -= 0.0005; }
  
  Vector3 max(x_max, y_max, z_max), min(x_min, y_min, z_min);
	
	box = aabb(min,max);
  
}

__host__ __device__ aabb Triangle::getBox() {
  return box;
}

__host__ __device__ long long Triangle::getMorton() {
  return morton_code;
}

__host__ __device__ void Triangle::setMorton(long long code) { 
  morton_code = code;
}

__host__ __device__ Vector3 Triangle::operator[](int i) const {
  if(i < 0 && i > 2) assert(0);
  return vertex[i];
}

__host__ __device__ Vector3& Triangle::operator[](int i) {
  if(i < 0 && i > 2) assert(0);
  return vertex[i];
}

__host__ __device__ Vector3 Triangle::getCentroid() {
  return centroid;
}

__host__ __device__ Material Triangle::getMaterial() {
  return mat_ptr;
}

__host__ __device__ void Triangle::resizeBoundingBox() {
  bounding_box(box);
}
