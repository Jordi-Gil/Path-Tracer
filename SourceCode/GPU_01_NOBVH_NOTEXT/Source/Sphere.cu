#include "hip/hip_runtime.h"
#include "Sphere.cuh"

__host__ __device__ Sphere::Sphere(Vector3 cen, float r, Material mat) { 
  center = cen;
  radius = r;
  mat_ptr = mat;
}

__device__ bool Sphere::hit(const Ray& r, float t_min, float t_max, hit_record &rec) {
  
  Vector3 oc = r.origin() - center;
  float a = dot(r.direction(), r.direction());
  float b = dot(oc, r.direction());
  float c = dot(oc, oc) - radius*radius;
  float discriminant = b*b - a*c;
  
  if(discriminant > 0){
    float temp = (-b - sqrt(b*b-a*c))/a;
    if(temp < t_max && temp > t_min){
      rec.t = temp;
      rec.point = r.point_at_parameter(rec.t);
      rec.normal = (rec.point - center) / radius;
      rec.mat_ptr = this->mat_ptr;
      return true;
    }
    
    temp = (-b + sqrt(b*b-a*c))/a;
    if(temp < t_max && temp > t_min){
      rec.t = temp;
      rec.point = r.point_at_parameter(rec.t);
      rec.normal = (rec.point - center) / radius;
      rec.mat_ptr = this->mat_ptr;
      return true;
    }
  }
  return false;
}

__host__ __device__ Vector3 Sphere::getCenter() {
    return center;
}

__host__ __device__ float Sphere::getRadius() {
    return radius;
}

__host__ __device__ Material Sphere::getMaterial() {
    return mat_ptr;
}
